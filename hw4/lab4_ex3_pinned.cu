
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <stdlib.h>

#define DataType double

// Compute C = A * B
__global__ void gemm(DataType *A, DataType *B, DataType *C, int numARows,
                      int numAColumns, int numBRows, int numBColumns){

  //@@ Insert code to implement matrix multiplication here
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    DataType sum = 0.0;

      if( col < numBColumns && row < numARows )
      {
        for(int m = 0; m < numAColumns; m++) 
        {
            sum += A[row * numAColumns + m] * B[m * numBColumns + col];
        }
        C[row * numBColumns + col] = sum;
      }

}

int main(int argc, char **argv) {
  
  DataType *hostA; // The A matrix
  DataType *hostB; // The B matrix
  DataType *hostC; // The output C matrix
  DataType *resultRef; // The reference result
  DataType *deviceA;
  DataType *deviceB;
  DataType *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;
  int numCColumns;

  //@@ Insert code below to read in numARows, numAColumns, numBColumns from args
  numARows = atoi(argv[1]);
  numAColumns = atoi(argv[2]);
  numBRows = atoi(argv[3]);
  numBColumns = atoi(argv[4]);
  numCRows = numARows;
  numCColumns = numBColumns;
  printf("Input matrix dim (%d x %d) (%d x %d) (%d x %d)\n", numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

  //@@ Insert code below to allocate Host memory for input and output
  hipHostMalloc((void**)&hostA, sizeof(DataType) * numARows * numAColumns, hipHostMallocDefault);
  hipHostMalloc((void**)&hostB, sizeof(DataType) * numBRows * numBColumns, hipHostMallocDefault);
  hipHostMalloc((void**)&hostC, sizeof(DataType) * numCRows * numCColumns, hipHostMallocDefault);
  resultRef = (DataType*) malloc(numCRows * numCColumns * sizeof(DataType));
  
  //@@ Insert code below to initialize hostA and hostB to random numbers, and create reference result in CPU
  for (int i = 0; i < numARows; i++) {
        for (int j = 0; j < numAColumns; j++) {
           DataType randn1 = rand() / (DataType) RAND_MAX;
           hostA[i*numAColumns + j] = randn1;
        }
  }
  for (int i = 0; i < numBRows; i++) {
        for (int j = 0; j < numBColumns; j++) {
           DataType randn2 = rand() / (DataType) RAND_MAX;
           hostB[i*numBColumns + j] = randn2;
        }
  }
  
  for (int i = 0; i < numARows; i++) {
        for (int j = 0; j < numBColumns; j++) {
          resultRef[i*numBColumns + j] = 0.0;
          for (int k = 0; k < numAColumns; k++) {
            resultRef[i*numBColumns + j] +=  hostA[i*numAColumns + k] * hostB[k*numBColumns + j];
          }        
        }
  }
   
  //@@ Insert code below to allocate GPU memory here
  hipMalloc((void**)&deviceA, numARows * numAColumns * sizeof(DataType));
  hipMalloc((void**)&deviceB, numBRows * numBColumns * sizeof(DataType));
  hipMalloc((void**)&deviceC, numCRows * numCColumns * sizeof(DataType));

  //@@ Insert code to below to Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(DataType), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(DataType), hipMemcpyHostToDevice);

  //@@ Initialize the grid and block dimensions here
  int Dbx = 32;
  int Dby = 32;
  int Dgx = (numCColumns + Dbx - 1) / Dbx;
  int Dgy = (numCRows + Dby - 1) / Dby;

  //@@ Launch the GPU Kernel here
  gemm<<<dim3(Dgx, Dgy, 1), dim3(Dbx, Dby, 1)>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns);

  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(DataType), hipMemcpyDeviceToHost);

  //@@ Insert code below to compare the output with the reference
  int is_equal = 1;
  for (int i = 0; i < numCRows; i++) {
    for (int j = 0; j < numCColumns; j++) {
      if (fabs(hostC[i*numCColumns + j] - resultRef[i*numCColumns + j]) > 1e-4)
        is_equal = 0;
        break;
    }       
  }
  if(is_equal == 0)
    printf("Result is not equal to reference");
  else
    printf("Result is equal to reference");

  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  //@@ Free the CPU memory here
  hipHostFree(hostA);
  hipHostFree(hostB);
  hipHostFree(hostC);
  free(resultRef);

  return 0;
}

