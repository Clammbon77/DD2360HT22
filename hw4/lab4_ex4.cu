#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <thrust/device_ptr.h>
#include <thrust/sequence.h>

#define gpuCheck(stmt)                                               \
  do {                                                               \
      hipError_t err = stmt;                                        \
      if (err != hipSuccess) {                                      \
          printf("ERROR. Failed to run stmt %s\n", #stmt);           \
          break;                                                     \
      }                                                              \
  } while (0)

// Macro to check the cuBLAS status
#define cublasCheck(stmt)                                            \
  do {                                                               \
      hipblasStatus_t err = stmt;                                     \
      if (err != HIPBLAS_STATUS_SUCCESS) {                            \
          printf("ERROR. Failed to run cuBLAS stmt %s\n", #stmt);    \
          break;                                                     \
      }                                                              \
  } while (0)

// Macro to check the cuSPARSE status
#define cusparseCheck(stmt)                                          \
  do {                                                               \
      hipsparseStatus_t err = stmt;                                   \
      if (err != HIPSPARSE_STATUS_SUCCESS) {                          \
          printf("ERROR. Failed to run cuSPARSE stmt %s\n", #stmt);  \
          break;                                                     \
      }                                                              \
  } while (0)


struct timeval t_start, t_end;
void cputimer_start(){
  gettimeofday(&t_start, 0);
}
void cputimer_stop(const char* info){
  gettimeofday(&t_end, 0);
  double time = (1000000.0*(t_end.tv_sec-t_start.tv_sec) + t_end.tv_usec-t_start.tv_usec);
  printf("Timing - %s. \t\tElasped %.0f microseconds \n", info, time);
}

// Initialize the sparse matrix needed for the heat time step
void matrixInit(double* A, int* ArowPtr, int* AcolIndx, int dimX,
    double alpha) {
  // Stencil from the finete difference discretization of the equation
  double stencil[] = { 1, -2, 1 };
  // Variable holding the position to insert a new element
  size_t ptr = 0;
  // Insert a row of zeros at the beginning of the matrix
  ArowPtr[1] = ptr;
  // Fill the non zero entries of the matrix
  for (int i = 1; i < (dimX - 1); ++i) {
    // Insert the elements: A[i][i-1], A[i][i], A[i][i+1]
    for (int k = 0; k < 3; ++k) {
      // Set the value for A[i][i+k-1]
      A[ptr] = stencil[k];
      // Set the column index for A[i][i+k-1]
      AcolIndx[ptr++] = i + k - 1;
    }
    // Set the number of newly added elements
    ArowPtr[i + 1] = ptr;
  }
  // Insert a row of zeros at the end of the matrix
  ArowPtr[dimX] = ptr;
}

int main(int argc, char **argv) {
  int device = 0;            // Device to be used
  int dimX;                  // Dimension of the metal rod
  int nsteps;                // Number of time steps to perform
  double alpha = 0.4;        // Diffusion coefficient
  double* temp;              // Array to store the final time step
  double* A;                 // Sparse matrix A values in the CSR format
  int* ARowPtr;              // Sparse matrix A row pointers in the CSR format
  int* AColIndx;             // Sparse matrix A col values in the CSR format
  int nzv;                   // Number of non zero values in the sparse matrix
  double* tmp;               // Temporal array of dimX for computations
  size_t bufferSize = 0;     // Buffer size needed by some routines
  void* buffer = nullptr;    // Buffer used by some routines in the libraries
  int concurrentAccessQ;     // Check if concurrent access flag is set
  double zero = 0;           // Zero constant
  double one = 1;            // One constant
  double norm;               // Variable for norm values
  double error;              // Variable for storing the relative error
  double tempLeft = 200.;    // Left heat source applied to the rod
  double tempRight = 300.;   // Right heat source applied to the rod
  hipblasHandle_t cublasHandle;      // cuBLAS handle
  hipsparseHandle_t cusparseHandle;  // cuSPARSE handle
  hipsparseSpMatDescr_t Adescriptor;   // Mat descriptor needed by cuSPARSE
  hipsparseDnVecDescr_t Tdescriptor;   
  hipsparseDnVecDescr_t Ydescriptor; 

  // Read the arguments from the command line
  dimX = atoi(argv[1]);
  nsteps = atoi(argv[2]);

  // Print input arguments
  printf("The X dimension of the grid is %d \n", dimX);
  printf("The number of time steps to perform is %d \n", nsteps);

  // Get if the hipDeviceAttributeConcurrentManagedAccess flag is set
  gpuCheck(hipDeviceGetAttribute(&concurrentAccessQ, hipDeviceAttributeConcurrentManagedAccess, device));

  // Calculate the number of non zero values in the sparse matrix. This number
  // is known from the structure of the sparse matrix
  nzv = 3 * dimX - 6;

  //@@ Insert the code to allocate the temp, tmp and the sparse matrix
  //@@ arrays using Unified Memory
  cputimer_start();
  gpuCheck(hipMallocManaged(&temp, dimX * sizeof(double)));
  gpuCheck(hipMallocManaged(&tmp, dimX * sizeof(double)));
  gpuCheck(hipMallocManaged(&A, nzv * sizeof(double)));
  gpuCheck(hipMallocManaged(&ARowPtr, (dimX + 1) * sizeof(int)));
  gpuCheck(hipMallocManaged(&AColIndx, nzv * sizeof(int)));

  cputimer_stop("Allocating device memory");

  // Check if concurrentAccessQ is non zero in order to prefetch memory
  if (concurrentAccessQ) {
    cputimer_start();
    //@@ Insert code to prefetch in Unified Memory asynchronously to CPU
    gpuCheck(hipMemPrefetchAsync(A, nzv * sizeof(double), hipCpuDeviceId));
    gpuCheck(hipMemPrefetchAsync(ARowPtr, (dimX + 1) * sizeof(int), hipCpuDeviceId));
    gpuCheck(hipMemPrefetchAsync(AColIndx, nzv * sizeof(int), hipCpuDeviceId));
    cputimer_stop("Prefetching GPU memory to the host");
  }

  // Initialize the sparse matrix
  cputimer_start();
  matrixInit(A, ARowPtr, AColIndx, dimX, alpha);
  cputimer_stop("Initializing the sparse matrix on the host");

  //Initiliaze the boundary conditions for the heat equation
  cputimer_start();
  memset(temp, 0, sizeof(double) * dimX);
  temp[0] = tempLeft;
  temp[dimX - 1] = tempRight;
  cputimer_stop("Initializing memory on the host");

  if (concurrentAccessQ) {
    cputimer_start();
    //@@ Insert code to prefetch in Unified Memory asynchronously to the GPU
    gpuCheck(hipMemPrefetchAsync(A, nzv * sizeof(double), device));
    gpuCheck(hipMemPrefetchAsync(ARowPtr, (dimX + 1) * sizeof(int), device));
    gpuCheck(hipMemPrefetchAsync(AColIndx, nzv * sizeof(int), device));
    cputimer_stop("Prefetching GPU memory to the device");
  }

  //@@ Insert code to create the cuBLAS handle
  cublasCheck(hipblasCreate(&cublasHandle));
  //@@ Insert code to create the cuSPARSE handle
  cusparseCheck(hipsparseCreate(&cusparseHandle));

  //@@ Insert code to set the cuBLAS pointer mode to HIPSPARSE_POINTER_MODE_HOST
  cublasCheck(hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_HOST));
  cusparseCheck(hipsparseSetPointerMode(cusparseHandle, HIPSPARSE_POINTER_MODE_HOST));

  //@@ Insert code to call cusparse api to create the mat descriptor used by cuSPARSE
  cusparseCheck(
    hipsparseCreateCsr(
        &Adescriptor, dimX, dimX, nzv, ARowPtr, AColIndx, A,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F)
  );
  
  cusparseCheck(hipsparseCreateDnVec(&Tdescriptor, dimX, temp, HIP_R_64F));
  cusparseCheck(hipsparseCreateDnVec(&Ydescriptor, dimX, tmp, HIP_R_64F));

  //@@ Insert code to call cusparse api to get the buffer size needed by the sparse matrix per
  //@@ vector (SMPV) CSR routine of cuSPARSE
  cusparseCheck(
    hipsparseSpMV_bufferSize(
        cusparseHandle, 
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &one, Adescriptor, Tdescriptor, &zero, Ydescriptor,
        HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize
    )
);
  

  //@@ Insert code to allocate the buffer needed by cuSPARSE
  gpuCheck(hipMalloc(&buffer, bufferSize));
  
  int timeSpent = -1;
  struct timeval t_start_smpv;
  struct timeval t_end_smpv;

  // Perform the time step iterations
  for (int it = 0; it < nsteps; ++it) {
    //@@ Insert code to call cusparse api to compute the SMPV (sparse matrix multiplication) for
    //@@ the CSR matrix using cuSPARSE. This calculation corresponds to:
    //@@ tmp = 1 * A * temp + 0 * tmp
    gettimeofday(&t_start_smpv, 0);
    cusparseCheck(
      hipsparseSpMV(
          cusparseHandle, 
          HIPSPARSE_OPERATION_NON_TRANSPOSE,
          &one, Adescriptor, Tdescriptor, &zero, Ydescriptor, 
          HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer
      )
    );
    // Skip the first 
    gettimeofday(&t_end_smpv, 0);
    if (timeSpent == -1) {
      timeSpent = 0;
    } else {
      timeSpent += (int)(1000000.0*(t_end_smpv.tv_sec-t_start_smpv.tv_sec) + t_end_smpv.tv_usec-t_start_smpv.tv_usec);
    }
    

    //@@ Insert code to call cublas api to compute the axpy routine using cuBLAS.
    //@@ This calculation corresponds to: temp = alpha * tmp + temp
    cublasCheck(
      hipblasDaxpy(
          cublasHandle, dimX, &alpha, tmp, 1, temp, 1
      )
    );

    //@@ Insert code to call cublas api to compute the norm of the vector using cuBLAS
    //@@ This calculation corresponds to: ||temp||
    cublasCheck(
      hipblasDnrm2(
          cublasHandle, dimX, temp, 1, &norm
      )
    ); 

    // If the norm of A*temp is smaller than 10^-4 exit the loop
    if (norm < 1e-4)
      break;
  }
  printf("Timing - SpMV. \t\tElasped %d microseconds \n", timeSpent);

  // Calculate the exact solution using thrust
  thrust::device_ptr<double> thrustPtr(tmp);
  thrust::sequence(thrustPtr, thrustPtr + dimX, tempLeft,
      (tempRight - tempLeft) / (dimX - 1));

  // Calculate the relative approximation error:
  one = -1;

  //@@ Insert the code to call cublas api to compute the difference between the exact solution
  //@@ and the approximation
  //@@ This calculation corresponds to: tmp = -temp + tmp
  cublasCheck(hipblasDaxpy(cublasHandle, dimX, &one, temp, 1, tmp, 1)); 

  //@@ Insert the code to call cublas api to compute the norm of the absolute error
  //@@ This calculation corresponds to: || tmp ||
  cublasCheck(hipblasDnrm2(cublasHandle, dimX, tmp, 1, &norm)); 

  error = norm;

  //@@ Insert the code to call cublas api to compute the norm of temp
  //@@ This calculation corresponds to: || temp ||
  cublasCheck(hipblasDnrm2(cublasHandle, dimX, temp, 1, &norm));

  // Calculate the relative error
  error = error / norm;
  printf("The relative error of the approximation is %f\n", error);

  //@@ Insert the code to destroy the mat descriptor
  cusparseCheck(hipsparseDestroySpMat(Adescriptor));
  cusparseCheck(hipsparseDestroyDnVec(Tdescriptor)); 
  cusparseCheck(hipsparseDestroyDnVec(Ydescriptor));

  //@@ Insert the code to destroy the cuSPARSE handle
  cusparseCheck(hipsparseDestroy(cusparseHandle));

  //@@ Insert the code to destroy the cuBLAS handle
  cublasCheck(hipblasDestroy(cublasHandle)); 

  //@@ Insert the code for deallocating memory
  gpuCheck(hipFree(temp));
  gpuCheck(hipFree(tmp));
  gpuCheck(hipFree(A));
  gpuCheck(hipFree(ARowPtr));
  gpuCheck(hipFree(AColIndx));
  gpuCheck(hipFree(buffer));

  return 0;
}
