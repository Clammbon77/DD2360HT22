

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <stdlib.h>

#define DataType double

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
  //@@ Insert code to implement vector addition here
  const int id = blockIdx.x * blockDim.x + threadIdx.x;

  if (id < len)
    out[id] = in1[id] + in2[id];
}

double cpuSecond() {
   struct timeval tp;
   gettimeofday(&tp,NULL);
   return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

int main(int argc, char **argv) {
  
  int inputLength;
  int S_seg;
  DataType *hostInput1;
  DataType *hostInput2;
  DataType *hostOutput;
  DataType *resultRef;
  DataType *deviceInput1;
  DataType *deviceInput2;
  DataType *deviceOutput;

  //@@ Insert code below to read in inputLength and segement size from args
  inputLength = atoi(argv[1]);
  S_seg = atoi(argv[2]);
  const int nStreams = inputLength / S_seg;
  printf("The input length is %d\n", inputLength);
  printf("The stream size is %d\n", S_seg);
  printf("The number of streams is %d\n", nStreams);
  
  //@@ Insert code below to allocate Host memory for input and output
  hostInput1 = (DataType*)malloc(inputLength*sizeof(DataType)); 
  hostInput2 = (DataType*)malloc(inputLength*sizeof(DataType)); 
  hostOutput = (DataType*)malloc(inputLength*sizeof(DataType));
  resultRef = (DataType*)malloc(inputLength*sizeof(DataType));
   
  // @@ Insert code below to create streams
  hipStream_t streams[nStreams]; 
  for(int i = 0; i < nStreams; i++) {
    hipStreamCreate(&streams[i]);
  }
      
  //@@ Insert code below to initialize hostInput1 and hostInput2 to random numbers, and create reference result in CPU
  for (int i = 0; i< inputLength; ++i) {
        DataType rand1 = rand() / (DataType) (RAND_MAX + 1.0); // Random number in interval [0, 1.0)
        DataType rand2 = rand() / (DataType) (RAND_MAX + 1.0); // Random number in interval [0, 1.0)
        hostInput1[i] = rand1;
        hostInput2[i] = rand2;
        resultRef[i] = rand1 + rand2;
    }
  
  //@@ Insert code below to allocate GPU memory here
  hipMalloc(&deviceInput1, inputLength*sizeof(DataType));
  hipMalloc(&deviceInput2, inputLength*sizeof(DataType));
  hipMalloc(&deviceOutput, inputLength*sizeof(DataType));
  
  //@@ Initialize the 1D grid and block dimensions here
  int Db = 256;
  int Dg = S_seg / Db; 

  //@@ Insert code to below to Copy mem to the GPU here
  for(int i = 0; i < nStreams; i++) {
        int offset = i * S_seg;
        hipMemcpyAsync(deviceInput1 + offset, hostInput1 + offset, S_seg * sizeof(DataType), 
                        hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(deviceInput2 + offset, hostInput2 + offset, S_seg * sizeof(DataType), 
                        hipMemcpyHostToDevice, streams[i]); 
        vecAdd<<<Dg, Db, 0, streams[i]>>>(deviceInput1 + offset, deviceInput2 + offset, deviceOutput + offset, S_seg);
        hipMemcpyAsync(hostOutput + offset, deviceOutput + offset, S_seg * sizeof(DataType), 
                        hipMemcpyDeviceToHost, streams[i]);
        hipStreamSynchronize(streams[i]);
  }
  
  for(int i = 0; i < nStreams; i++) {
    hipStreamDestroy(streams[i]);
  }

  hipDeviceSynchronize();
  
  //@@ Insert code below to compare the output with the reference 
  int is_equal = 1;
  for(int i = 0; i < inputLength; i++){
    if (hostOutput[i] - resultRef[i]> 1e-10)
      is_equal = 0;
    break;           
  }
  if(is_equal == 1)
    printf("Result is equal to reference");
  else
    printf("Result is not equal to reference");

  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);
  
  //@@ Free the CPU memory here
  free(hostInput1);
  free(hostInput2);
  free(hostOutput);
  free(resultRef);

  return 0;
}
